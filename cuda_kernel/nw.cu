/* Needleman Wunsch Algorithm Parallel Implementation */ 
#include <fstream>
#include "nw.h"


int main(int argc , char **argv)
{


        //size_t len;
        // Sequences to be aligned
        char  *seq_1;  // =  argv[ 1 ] ;
        char  *seq_2;  // =  argv[ 2 ] ;

        //FILE  *file1 , *file2 ;
        int   size1 , size2 ;
        ifstream file1("s1_400.txt");
        if (file1==NULL)
             perror ("Error opening file 1");
        else
        {
            file1.seekg (0, ios::end);
            size1 = file1.tellg();
            file1.seekg (0, ios::beg);

            seq_1 = (char *) malloc( 2 *size1 * sizeof(char));
            
            file1 >> seq_1 ;
            
            file1.close();
            printf("Seq 1: %s size1: %d\n",seq_1,size1);
        }
        

        ifstream file2("s2_400.txt");
        if (file2==NULL)
             perror ("Error opening file 2");
        else
        {
            file2.seekg (0, ios::end);
            size2 = file2.tellg();
            file2.seekg (0, ios::beg);
            seq_2 = (char *) malloc( 2* size2 * sizeof(char));
            
            file2 >> seq_2 ;
            
            file2.close();
            printf("Seq 2: %s size2: %d\n",seq_2,size2);
        }
        

        // Aligned sequences
        char  *seq_1_al;
        char  *seq_2_al;

    seq_1_al = (char *) malloc( 2 * size1 * sizeof(char));
    seq_2_al = (char *) malloc( 2 * size2 * sizeof(char));


    strcpy(seq_1_al,"");
    strcpy(seq_2_al,"");




    struct timespec t1,t2; double dt1;
    clock_gettime(CLOCK_REALTIME,  &t1);

    // Get alignment
    nw( seq_1, seq_2, seq_1_al, seq_2_al) ;   


    clock_gettime(CLOCK_REALTIME,  &t2);
    dt1 = (t2.tv_sec - t1.tv_sec) + (double) (t2.tv_nsec - t1.tv_nsec) * 1e-9;
    double time=dt1*1000 ;
    printf("\n%10f  kernel Time elapsed with only threads\n", time);

 /*  printf("\nOriginal Sequences:");
    printf("\n\t\t\t%s \n\t\t\t%s\n",seq_1,seq_2);
*/
    printf("\nAfter Alignment:");
    printf("\n%s \n%s\n\n",seq_1_al,seq_2_al);

return 0;
}
